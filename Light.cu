#include "hip/hip_runtime.h"
#include "framework.h"

//__device__ void Light::move_pixel_gpu(Point2<int> size, Point2<float>* setMat, Point2<float>* fromMat, float* weights) {
//__global__ void move_pixel_gpu(Point2<float>* setMat, Point2<float>* fromMat) {
//	return;
//	//float newHeg = 0;
//	//int offset = 0;
//	//float border = size.x / 10;
//	//float setVal = 1;
//	//for (int x = 0; x < size.x; x++) {
//	//	for (int y = 0; y < size.y; y++) {
//	//		for (int k = 0; k < 3; k++) {
//	//			setVal = 1;
//	//			offset = get_offset(x - 1, y);
//	//			newHeg = offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
//	//			offset = get_offset(x, y + 1);
//	//			newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
//	//			offset = get_offset(x + 1, y);
//	//			newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
//	//			offset = get_offset(x, y - 1);
//	//			newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
//	//			newHeg /= 4;
//	//			if (x<border || y<border || x>size.x - border || y>size.y - border) {
//	//				if (x < border)
//	//					setVal = x / border;
//	//				if (y < border)
//	//					setVal = y / border;
//	//				if (x > size.x - border)
//	//					setVal = (size.x - x) / border;
//	//				if (y > size.y - border)
//	//					setVal = (size.y - y) / border;
//	//				//weights[get_offset(x, y) + k] = setVal;
//	//				setVal = -(pow((setVal - 1), 2)) + 1;
//	//			}
//	//			setMat[get_offset(x, y) + k].y = (fromMat[get_offset(x, y) + k].y + (newHeg - fromMat[get_offset(x, y) + k].x * weights[get_offset(x, y) + k]));
//	//			setMat[get_offset(x, y) + k].x = 0.999 * setVal * (fromMat[get_offset(x, y) + k].x + setMat[get_offset(x, y) + k].y);
//	//		}
//	//	}
//	//}
//
//}

__device__ int get_offset(int x, int y, Point2<int> size) {
	if (x < 0 || x >= size.x || y < 0 || y >= size.y)
		return -1;
	return y * size.x * 3 + x * 3;
}

__global__ void move_pixel_gpu(Point2<int> size, Point2<float>* fromMat, Point2<float>* setMat, float* weights, int* img, int segment, float* imgAccum) {

	float newHeg = 0;
	int offset = 0;
	float border = size.x / 10;
	float setVal = 1;/*
	for (int x = 0; x < size.x; x++) {
		for (int y = 0; y < size.y; y++) {*/
	int x, y;
	//x = 0; y = 0;
	x = (blockIdx.x * GPU_THREADS + threadIdx.x) % size.x;
	y = (blockIdx.x * GPU_THREADS + threadIdx.x) / size.x;
	if (x >= size.x || y >= size.y) return;
	/*for (int i = 0; i < 3; i++) {
		setMat[get_offset(x,y,size) + i].x = 255;
	}*/
	//setMat[get_offset(x, y, size) + 0].x = threadIdx.x/2;
	//setMat[get_offset(x, y, size) + 1].x = blockIdx.x*20;
	//setMat[get_offset(x, y, size) + 2].x = get_offset(x, y, size) == -1 ? 255: 0;

	static int iddd = 0;
	iddd += 1;

	for (int k = 0; k < 3; k++) {
		setVal = 1;
		offset = get_offset(x - 1, y, size);
		newHeg = offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
		offset = get_offset(x, y + 1, size);
		newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
		offset = get_offset(x + 1, y, size);
		newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
		offset = get_offset(x, y - 1, size);
		newHeg += offset >= 0 ? fromMat[offset + k].x * weights[offset + k] : 0;
		newHeg /= 4;
		if (x<border || y<border || x>size.x - border || y>size.y - border) {
			if (x < border)
				setVal = x / border;
			if (y < border)
				setVal = y / border;
			if (x > size.x - border)
				setVal = (size.x - x) / border;
			if (y > size.y - border)
				setVal = (size.y - y) / border;
			//weights[get_offset(x, y) + k] = setVal;
			setVal = -(pow((setVal - 1), 2)) + 1;
		}
		setMat[get_offset(x, y, size) + k].y = (fromMat[get_offset(x, y, size) + k].y + (newHeg - fromMat[get_offset(x, y, size) + k].x * weights[get_offset(x, y, size) + k]));
		setMat[get_offset(x, y, size) + k].x = 0.999 * setVal * (fromMat[get_offset(x, y, size) + k].x + setMat[get_offset(x, y, size) + k].y);
		imgAccum[get_offset(x, y, size) + k] *= 0.995;
		imgAccum[get_offset(x, y, size) + k] += setMat[get_offset(x, y, size) + k].x>0? setMat[get_offset(x, y, size) + k].x : 0;
	}


	//create output image

	int heg = 0, hegOut = 0;
	//heg = (int)fromMat[get_offset(x, y) + 0].x + 127 * weights[get_offset(x,y) + 0] ;
	//heg = (int)setMat[get_offset(x, y, size) + 0].x;
	heg = (int)imgAccum[get_offset(x, y, size) + 0] - 64 * (weights[get_offset(x,y,size)+0]-1);
	if (heg > 255) heg = 255;
	if (heg < 0) heg = 0;
	hegOut = hegOut | (heg << 16);

	//heg = (int)fromMat[get_offset(x, y) + 1].x + 127 * weights[get_offset(x,y) + 1] ;
	//heg = (int)setMat[get_offset(x, y, size) + 1].x;
	heg = (int)imgAccum[get_offset(x, y, size) + 1] - 64 * (weights[get_offset(x,y,size)+1]-1);
	if (heg > 255) heg = 255;
	if (heg < 0) heg = 0;
	hegOut = hegOut | (heg << 8);

	//heg = (int)fromMat[get_offset(x, y) + 2].x + 127 * weights[get_offset(x,y) + 2] ;
	//heg = (int)setMat[get_offset(x, y, size) + 2].x;
	heg = (int)imgAccum[get_offset(x, y, size) + 2] - 64 * (weights[get_offset(x,y,size)+2]-1);
	if (heg > 255) heg = 255;
	if (heg < 0) heg = 0;
	hegOut = hegOut | (heg << 0);

	for (int sx = 0; sx < segment; sx++) {
		for (int sy = 0; sy < segment; sy++) {
			int offset = 0;
			img[y * size.x * segment * segment + sy * size.x * segment + x * segment + sx] = hegOut;
		}
	}

}

__host__ void move_gpu(Light& l, int* img) {
	Point2<float>* setMat, * fromMat;
	if (l.editingMat == 0)
		setMat = l.mat0, fromMat = l.mat1;
	else
		setMat = l.mat1, fromMat = l.mat0;

	hipMemcpy(l.gpuMat0, fromMat, l.size.x * l.size.y * 3 * sizeof(Point2<float>), hipMemcpyHostToDevice);
	hipMemcpy(l.gpuWeights, l.weights, l.size.x * l.size.y * 3 * sizeof(float), hipMemcpyHostToDevice);

	//move_pixel_gpu << < 1, 1 >> > (l.gpuMat0, l.gpuMat1);
	move_pixel_gpu << < (l.size.x * l.size.y / GPU_THREADS) + 1, GPU_THREADS  >> > (l.size, l.gpuMat0, l.gpuMat1, l.gpuWeights, l.gpuImg, l.segment, l.gpuImgAccum);

	hipMemcpy(setMat, l.gpuMat1, l.size.x * l.size.y * 3 * sizeof(Point2<float>), hipMemcpyDeviceToHost);
	hipMemcpy(img, l.gpuImg, l.pixSize.x * l.pixSize.y * sizeof(int), hipMemcpyDeviceToHost);

	l.editingMat = !l.editingMat;
	return;
}
